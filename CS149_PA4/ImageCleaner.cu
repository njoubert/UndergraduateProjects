#include <assert.h>
#include "hip/hip_runtime.h"

#include <stdio.h>

/*****************************************************/
/* CS149: ALL OF YOUR CODE SHOULD GO IN THIS FILE    */
/*****************************************************/

// You can modify these parameters to match the image input size
#define SIZEX    512
#define SIZEY    512

// no need to modify this:
#define BLOCK_SIZE 512
#define PI	3.14159256

__global__ void gpu_fftx(float *dReal, float *dImag, int size_x, int size_y) {

	//responsible for a single output cell in the FFT'd image.
	//runs in 512-wide threadblocks all working together

	int myRow = blockIdx.y; //processed by the whole threadblock
	int myCol = threadIdx.x + blockIdx.x * blockDim.x; //i calculate this result

	// Compute the value for this index
	float real_value = 0;
	float imag_value = 0;
	float term, realTerm, imagTerm;
	for (unsigned int n = 0; n < size_y; n++) { 				//(a+bi)(c+di) = (ac-bd) + (bc+ad)i
		term = -2 * PI * myCol * n / size_y;
		realTerm = term;//cos(term);
		imagTerm = term;//sin(term);

		real_value += (dReal[myRow * size_x + n] * realTerm)
				- (dImag[myRow * size_x + n] * imagTerm);

		imag_value += (dImag[myRow * size_x + n] * realTerm)
				+ (dReal[myRow * size_x + n] * imagTerm);

	}

	// Write the values back into the temporary buffer
	dReal[myRow * size_x + myCol] = real_value;
	dImag[myRow * size_x + myCol] = imag_value;

}
__global__ void gpu_ifftx(float *real_image, float *imag_image, int size_x, int size_y) {
  // Currently does nothing
}
__global__ void gpu_ffty(float *real_image, float *imag_image, int size_x, int size_y) {
  // Currently does nothing
}
__global__ void gpu_iffty(float *real_image, float *imag_image, int size_x, int size_y) {
  // Currently does nothing
}
__global__ void gpu_filter(float *real_image, float *imag_image, int size_x, int size_y) {
  // Currently does nothing
}

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Create a stream and initialize it
  hipStream_t filterStream;
  hipStreamCreate(&filterStream);

  // Alloc space on the device
  float *device_real, *device_imag;
  hipMalloc((void**)&device_real, matSize);
  hipMalloc((void**)&device_imag, matSize);

  // Start timing for transfer down
  hipEventRecord(start,filterStream);
  
  //-----------------------------------------------------------------
  // HERE IS WHERE WE COPY THE MATRICES DOWN TO THE DEVICE
  //-----------------------------------------------------------------
  hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice);
  hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice);
  
  // Stop timing for transfer down
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&transferDown,start,stop);

  // Start timing for the execution
  hipEventRecord(start,filterStream);



 
  //----------------------------------------------------------------
  // CS149: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual
  //    4. Stream to execute kernel on, should always be 'filterStream' for this project
  //
  // Also note that you pass the pointers to the device memory to the kernel call

  // we have to run kernels that does an fft.
  //	ffty:
  //		for each row in the image
  // 			for each cell in the row
  //				access all the other cells in the row


  //Here is how we split this up:
  // 	each threadblock is responsible for 512 elements in a row.
  //	threadblocks are arranged in a grid of 512-groups elements by all the rows.
  //	input image is SIZEX by SIZEY
  //	block this into a (SIZEX/512) by SIZEY grid
  //	each threadblock consists of 512 threads for a row in the grid.




  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(ceil((float)SIZEX/BLOCK_SIZE),SIZEY);
  printf("  Launching kernel with %d threads per block arranged in a grid of %dx%d.\n", dimBlock.x, dimGrid.x, dimGrid.y);
  gpu_fftx<<<dimGrid, dimBlock>>>(device_real,device_imag,size_x,size_y);







  // Finish timimg for the execution 
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&execution,start,stop);

  // Check for any errors in kernel launch
  hipError_t kernelResult = hipGetLastError();
  if( kernelResult != hipSuccess )
  {
    printf("WARNING: CUDA KERNEL CALL FAILED!  %s\n", hipGetErrorString(kernelResult));
  }

  // Start timing for the transfer up
  hipEventRecord(start,filterStream);

  //----------------------------------------------------------------
  // HERE IS WHERE WE COPY THE MATRICES BACK FROM THE DEVICE
  //----------------------------------------------------------------
  hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost);
  hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost);

  // Finish timing for transfer up
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&transferUp,start,stop);

  // Synchronize the stream
  hipStreamSynchronize(filterStream);
  // Destroy the stream
  hipStreamDestroy(filterStream);
  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Free the memory
  hipFree(device_real);
  hipFree(device_imag);

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

