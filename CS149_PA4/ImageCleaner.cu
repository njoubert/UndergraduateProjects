#include <assert.h>
#include "hip/hip_runtime.h"

#include <stdio.h>

/*****************************************************/
/* CS149: ALL OF YOUR CODE SHOULD GO IN THIS FILE    */
/*****************************************************/

// You can modify these parameters to match the image input size
#define SIZEX    512
#define SIZEY    512

// no need to modify this:
#define BLOCK_SIZE 128
#define PI	3.14159256f

__global__ void gpu_fftx(float *dReal, float *dImag, int size_x, int size_y) {

	//responsible for a single output cell in the FFT'd image.
	//runs in 512-wide threadblocks all working together

	int myRow = blockIdx.y; //processed by the whole threadblock
	int myCol = threadIdx.x + blockIdx.x * blockDim.x; //i calculate this result

	// Compute the value for this index
	float real_value = 0;
	float imag_value = 0;
	float term, realTerm, imagTerm;
	for (unsigned int n = 0; n < size_y; n++) { 				//(a+bi)(c+di) = (ac-bd) + (bc+ad)i
		term = -2 * PI * myCol * n / size_y;
		realTerm = cos(term);
		imagTerm = sin(term);

		real_value += (dReal[myRow * size_x + n] * realTerm)
				- (dImag[myRow * size_x + n] * imagTerm);

		imag_value += (dImag[myRow * size_x + n] * realTerm)
				+ (dReal[myRow * size_x + n] * imagTerm);

	}

	// Write the values back into the temporary buffer
	dReal[myRow * size_x + myCol] = real_value;
	dImag[myRow * size_x + myCol] = imag_value;

}
__global__ void gpu_ifftx(float *dReal, float *dImag, int size_x, int size_y) {

	//responsible for a single output cell in the FFT'd image.
	//runs in 512-wide threadblocks all working together

	int myRow = blockIdx.y; //processed by the whole threadblock
	int myCol = threadIdx.x + blockIdx.x * blockDim.x; //i calculate this result

	// Compute the value for this index
	float real_value = 0;
	float imag_value = 0;
	float term, realTerm, imagTerm;
	for (unsigned int n = 0; n < size_y; n++) { 				//(a+bi)(c+di) = (ac-bd) + (bc+ad)i
		term = 2 * PI * myCol * n / size_y;
		realTerm = cos(term);
		imagTerm = sin(term);

		real_value += (dReal[myRow * size_x + n] * realTerm)
				- (dImag[myRow * size_x + n] * imagTerm);

		imag_value += (dImag[myRow * size_x + n] * realTerm)
				+ (dReal[myRow * size_x + n] * imagTerm);

	}

	// Write the values back into the temporary buffer
	dReal[myRow * size_x + myCol] = real_value / size_y;
	dImag[myRow * size_x + myCol] = imag_value / size_y;


}
__global__ void gpu_ffty(float *dReal, float *dImag, int size_x, int size_y) {
	int myRow = blockIdx.y * blockDim.y + threadIdx.y;
	int myCol = blockIdx.x;

	//runs over elements in column
	// Compute the value for this index
	float real_value = 0;
	float imag_value = 0;
	float term, realTerm, imagTerm;
	for (unsigned int n = 0; n < size_x; n++) { 				//(a+bi)(c+di) = (ac-bd) + (bc+ad)i
		term = -2 * PI * myRow * n / size_x;
		realTerm = cos(term);
		imagTerm = sin(term);

		real_value += (dReal[n * size_x + myCol] * realTerm)
				- (dImag[n * size_x + myCol] * imagTerm);

		imag_value += (dImag[n * size_x + myCol] * realTerm)
				+ (dReal[n * size_x + myCol] * imagTerm);
	}

	// Write the values back into the temporary buffer
	dReal[myRow * size_x + myCol] = real_value;
	dImag[myRow * size_x + myCol] = imag_value;

}
__global__ void gpu_iffty(float *dReal, float *dImag, int size_x, int size_y) {
	int myRow = blockIdx.y * blockDim.y + threadIdx.y;
	int myCol = blockIdx.x;

	//runs over elements in column
	// Compute the value for this index
	float real_value = 0;
	float imag_value = 0;
	float term, realTerm, imagTerm;
	for (unsigned int n = 0; n < size_x; n++) { 				//(a+bi)(c+di) = (ac-bd) + (bc+ad)i
		term = 2 * PI * myRow * n / size_x;
		realTerm = cos(term);
		imagTerm = sin(term);

		real_value += (dReal[n * size_x + myCol] * realTerm)
				- (dImag[n * size_x + myCol] * imagTerm);

		imag_value += (dImag[n * size_x + myCol] * realTerm)
				+ (dReal[n * size_x + myCol] * imagTerm);
	}

	// Write the values back into the temporary buffer
	dReal[myRow * size_x + myCol] = real_value / size_x;
	dImag[myRow * size_x + myCol] = imag_value / size_x;
}
__global__ void gpu_filter(float *dReal, float *dImag, int size_x, int size_y) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int eightX = size_x / 8;
	//int eight7X = size_x - eightX;
	int eightY = size_y / 8;
	int eight7Y = size_y - eightY;

	if (!(x < eightX && y < eightY) && !(x < eightX && y >= eight7Y)
			&& !(x >= eight7Y && y < eightY) && !(x >= eight7Y && y
			>= eight7Y)) {
		dReal[y * size_x + x] = 0;
		dImag[y * size_x + x] = 0;
	}

}

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Create a stream and initialize it
  hipStream_t filterStream;
  hipStreamCreate(&filterStream);

  // Alloc space on the device
  float *device_real, *device_imag;
  hipMalloc((void**)&device_real, matSize);
  hipMalloc((void**)&device_imag, matSize);

  // Start timing for transfer down
  hipEventRecord(start,filterStream);
  
  //-----------------------------------------------------------------
  // HERE IS WHERE WE COPY THE MATRICES DOWN TO THE DEVICE
  //-----------------------------------------------------------------
  hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice);
  hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice);
  
  // Stop timing for transfer down
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&transferDown,start,stop);

  // Start timing for the execution
  hipEventRecord(start,filterStream);



 
  //----------------------------------------------------------------
  // CS149: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual
  //    4. Stream to execute kernel on, should always be 'filterStream' for this project
  //
  // Also note that you pass the pointers to the device memory to the kernel call

  // we have to run kernels that does an fft.
  //	ffty:
  //		for each row in the image
  // 			for each cell in the row
  //				access all the other cells in the row


  // FOR ROW-BASED FFT (fftx and ifftx)
  //Here is how we split this up:
  // 	each threadblock is responsible for 512 elements in a row.
  //	threadblocks are arranged in a grid of 512-groups elements by all the rows.
  //	input image is SIZEX by SIZEY
  //	block this into a (SIZEX/512) by SIZEY grid
  //	each threadblock consists of 512 threads for a row in the grid.

  dim3 fftx_dimBlock(BLOCK_SIZE,1);
  dim3 fftx_dimGrid((int)ceil((float)SIZEX/BLOCK_SIZE),SIZEY);

  dim3 ffty_dimBlock(1,BLOCK_SIZE);
  dim3 ffty_dimGrid(SIZEX, (int)ceil((float)SIZEY/BLOCK_SIZE));

  dim3 filter_dimBlock(16,16);
  dim3 filter_dimGrid((int)ceil((float)SIZEX/16), (int)ceil((float)SIZEY/16));


  printf("  Launching fftx kernel with %d threads per block arranged in a grid of %dx%d.\n", fftx_dimBlock.x, fftx_dimGrid.x, fftx_dimGrid.y);
  gpu_fftx<<<fftx_dimGrid, fftx_dimBlock, 0, filterStream>>>(device_real,device_imag,size_x,size_y);
  printf("  Launching fftx kernel with %d threads per block arranged in a grid of %dx%d.\n", ffty_dimBlock.y, ffty_dimGrid.x, ffty_dimGrid.y);
  gpu_ffty<<<ffty_dimGrid, ffty_dimBlock, 0, filterStream>>>(device_real,device_imag,size_x,size_y);


  printf("  Launching filter kernel with %dx%d threads per block arranged in a grid of %dx%d.\n", filter_dimBlock.x,filter_dimBlock.y, filter_dimGrid.x, filter_dimGrid.y);
  gpu_filter<<<filter_dimGrid, filter_dimBlock, 0, filterStream>>>(device_real,device_imag,size_x,size_y);


  printf("  Launching ifftx kernel with %d threads per block arranged in a grid of %dx%d.\n", fftx_dimBlock.x, fftx_dimGrid.x, fftx_dimGrid.y);
  gpu_ifftx<<<fftx_dimGrid, fftx_dimBlock, 0, filterStream>>>(device_real,device_imag,size_x,size_y);
  printf("  Launching fftx kernel with %d threads per block arranged in a grid of %dx%d.\n", ffty_dimBlock.y, ffty_dimGrid.x, ffty_dimGrid.y);
  gpu_iffty<<<ffty_dimGrid, ffty_dimBlock, 0, filterStream>>>(device_real,device_imag,size_x,size_y);

  // Finish timimg for the execution 
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&execution,start,stop);

  // Check for any errors in kernel launch
  hipError_t kernelResult = hipGetLastError();
  if( kernelResult != hipSuccess )
  {
    printf("WARNING: CUDA KERNEL CALL FAILED!  %s\n", hipGetErrorString(kernelResult));
  }

  // Start timing for the transfer up
  hipEventRecord(start,filterStream);

  //----------------------------------------------------------------
  // HERE IS WHERE WE COPY THE MATRICES BACK FROM THE DEVICE
  //----------------------------------------------------------------
  hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost);
  hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost);

  // Finish timing for transfer up
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&transferUp,start,stop);

  // Synchronize the stream
  hipStreamSynchronize(filterStream);
  // Destroy the stream
  hipStreamDestroy(filterStream);
  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Free the memory
  hipFree(device_real);
  hipFree(device_imag);

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

