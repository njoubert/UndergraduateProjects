#include <assert.h>
#include "hip/hip_runtime.h"

#include <stdio.h>

/*****************************************************/
/* CS149: ALL OF YOUR CODE SHOULD GO IN THIS FILE    */
/*****************************************************/

// You can modify these parameters to match the image input size
#define SIZEX    512
#define SIZEY    512

__global__ void exampleKernel(float *real_image, float *imag_image, int size_x, int size_y)
{
  // Currently does nothing
}

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Create a stream and initialize it
  hipStream_t filterStream;
  hipStreamCreate(&filterStream);

  // Alloc space on the device
  float *device_real, *device_imag;
  hipMalloc((void**)&device_real, matSize);
  hipMalloc((void**)&device_imag, matSize);

  // Start timing for transfer down
  hipEventRecord(start,filterStream);
  
  //-----------------------------------------------------------------
  // HERE IS WHERE WE COPY THE MATRICES DOWN TO THE DEVICE
  //-----------------------------------------------------------------
  hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice);
  hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice);
  
  // Stop timing for transfer down
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&transferDown,start,stop);

  // Start timing for the execution
  hipEventRecord(start,filterStream);



 
  //----------------------------------------------------------------
  // CS149: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual
  //    4. Stream to execute kernel on, should always be 'filterStream' for this project
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  exampleKernel<<<1,128,0,filterStream>>>(device_real,device_imag,size_x,size_y);







  // Finish timimg for the execution 
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&execution,start,stop);

  // Check for any errors in kernel launch
  hipError_t kernelResult = hipGetLastError();
  if( kernelResult != hipSuccess )
  {
    printf("WARNING: CUDA KERNEL CALL FAILED!  %s\n", hipGetErrorString(kernelResult));
  }

  // Start timing for the transfer up
  hipEventRecord(start,filterStream);

  //----------------------------------------------------------------
  // HERE IS WHERE WE COPY THE MATRICES BACK FROM THE DEVICE
  //----------------------------------------------------------------
  hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost);
  hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost);

  // Finish timing for transfer up
  hipEventRecord(stop,filterStream);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&transferUp,start,stop);

  // Synchronize the stream
  hipStreamSynchronize(filterStream);
  // Destroy the stream
  hipStreamDestroy(filterStream);
  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Free the memory
  hipFree(device_real);
  hipFree(device_imag);

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

